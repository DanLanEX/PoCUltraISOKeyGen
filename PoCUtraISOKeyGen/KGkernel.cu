#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include <stdio.h>
#define m_res  0x58F0709D5591682F
typedef unsigned long long ull;

__device__ __inline__ void add_mod(ull &a, ull b, ull m)
{
    b = a + b;
    if (b < a)
    {
        a = (b%m + m_res) % m;
    }
    else
    {
        a = b%m;
    }
}

__device__ __inline__ void  mul_mod(ull &a, ull b, ull m)
{
    ull tmp = a%m;
    a = 0;
    while (b)
    {
        if (b & 1)
            add_mod(a, tmp, m);

        add_mod(tmp, tmp, m);
        b >>= 1;
    }
}

__device__ __inline__ ull exp_mod(ull c, ull e, ull n)
{
    c %= n;
    ull result = 1;
    while (e > 0)
    {
        if (e & 1)
        {
            mul_mod(result, c, n);//result = (result*c) % n;
        }
        e >>= 1;
        mul_mod(c, c, n);//c = (c*c) % n;
    }

    return result;
}



typedef unsigned long long ull;
typedef unsigned int uint;
typedef unsigned char uchar;
#define shift(x, n) (((x) << (n)) | ((x) >> (32-(n))))
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))    
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

#define FF(a ,b ,c ,d ,Mj ,s ,ti)  a = b + (shift((a + F(b, c, d) + Mj + ti) , s))
#define GG(a, b, c, d, Mj, s, ti)  a = b + (shift((a + G(b, c, d) + Mj + ti) , s))
#define HH(a, b, c, d, Mj, s, ti)  a = b + (shift((a + H(b, c, d) + Mj + ti) , s))
#define II(a, b, c, d, Mj, s, ti)  a = b + (shift((a + I(b, c, d) + Mj + ti) , s))

#define A 0x67452301
#define B 0xefcdab89
#define C 0x98badcfe
#define D 0x10325476

//msg must be padded 512 bit.
__device__ __inline__ ull md5_512(uint* msg)
{
    uint a = A, b = B, c = C, d = D;

    FF(a, b, c, d, msg[0], 7, 0xd76aa478);
    FF(d, a, b, c, msg[1], 12, 0xe8c7b756);
    FF(c, d, a, b, msg[2], 17, 0x242070db);
    FF(b, c, d, a, msg[3], 22, 0xc1bdceee);
    FF(a, b, c, d, msg[4], 7, 0xf57c0faf);
    FF(d, a, b, c, msg[5], 12, 0x4787c62a);
    FF(c, d, a, b, msg[6], 17, 0xa8304613);
    FF(b, c, d, a, msg[7], 22, 0xfd469501);
    FF(a, b, c, d, msg[8], 7, 0x698098d8);
    FF(d, a, b, c, msg[9], 12, 0x8b44f7af);
    FF(c, d, a, b, msg[10], 17, 0xffff5bb1);
    FF(b, c, d, a, msg[11], 22, 0x895cd7be);
    FF(a, b, c, d, msg[12], 7, 0x6b901122);
    FF(d, a, b, c, msg[13], 12, 0xfd987193);
    FF(c, d, a, b, msg[14], 17, 0xa679438e);
    FF(b, c, d, a, msg[15], 22, 0x49b40821);

    GG(a, b, c, d, msg[1], 5, 0xf61e2562);
    GG(d, a, b, c, msg[6], 9, 0xc040b340);
    GG(c, d, a, b, msg[11], 14, 0x265e5a51);
    GG(b, c, d, a, msg[0], 20, 0xe9b6c7aa);
    GG(a, b, c, d, msg[5], 5, 0xd62f105d);
    GG(d, a, b, c, msg[10], 9, 0x02441453);
    GG(c, d, a, b, msg[15], 14, 0xd8a1e681);
    GG(b, c, d, a, msg[4], 20, 0xe7d3fbc8);
    GG(a, b, c, d, msg[9], 5, 0x21e1cde6);
    GG(d, a, b, c, msg[14], 9, 0xc33707d6);
    GG(c, d, a, b, msg[3], 14, 0xf4d50d87);
    GG(b, c, d, a, msg[8], 20, 0x455a14ed);
    GG(a, b, c, d, msg[13], 5, 0xa9e3e905);
    GG(d, a, b, c, msg[2], 9, 0xfcefa3f8);
    GG(c, d, a, b, msg[7], 14, 0x676f02d9);
    GG(b, c, d, a, msg[12], 20, 0x8d2a4c8a);

    HH(a, b, c, d, msg[5], 4, 0xfffa3942);
    HH(d, a, b, c, msg[8], 11, 0x8771f681);
    HH(c, d, a, b, msg[11], 16, 0x6d9d6122);
    HH(b, c, d, a, msg[14], 23, 0xfde5380c);
    HH(a, b, c, d, msg[1], 4, 0xa4beea44);
    HH(d, a, b, c, msg[4], 11, 0x4bdecfa9);
    HH(c, d, a, b, msg[7], 16, 0xf6bb4b60);
    HH(b, c, d, a, msg[10], 23, 0xbebfbc70);
    HH(a, b, c, d, msg[13], 4, 0x289b7ec6);
    HH(d, a, b, c, msg[0], 11, 0xeaa127fa);
    HH(c, d, a, b, msg[3], 16, 0xd4ef3085);
    HH(b, c, d, a, msg[6], 23, 0x04881d05);
    HH(a, b, c, d, msg[9], 4, 0xd9d4d039);
    HH(d, a, b, c, msg[12], 11, 0xe6db99e5);
    HH(c, d, a, b, msg[15], 16, 0x1fa27cf8);
    HH(b, c, d, a, msg[2], 23, 0xc4ac5665);

    II(a, b, c, d, msg[0], 6, 0xf4292244);
    II(d, a, b, c, msg[7], 10, 0x432aff97);
    II(c, d, a, b, msg[14], 15, 0xab9423a7);
    II(b, c, d, a, msg[5], 21, 0xfc93a039);
    II(a, b, c, d, msg[12], 6, 0x655b59c3);
    II(d, a, b, c, msg[3], 10, 0x8f0ccc92);
    II(c, d, a, b, msg[10], 15, 0xffeff47d);
    II(b, c, d, a, msg[1], 21, 0x85845dd1);
    II(a, b, c, d, msg[8], 6, 0x6fa87e4f);
    II(d, a, b, c, msg[15], 10, 0xfe2ce6e0);
    II(c, d, a, b, msg[6], 15, 0xa3014314);
    II(b, c, d, a, msg[13], 21, 0x4e0811a1);
    II(a, b, c, d, msg[4], 6, 0xf7537e82);
    II(d, a, b, c, msg[11], 10, 0xbd3af235);
    II(c, d, a, b, msg[2], 15, 0x2ad7d2bb);
    II(b, c, d, a, msg[9], 21, 0xeb86d391);

    a = a + A;
    b = b + B;
    //c = c + C;
    //d = d + D;
    return (static_cast<ull>(b) << 32) | a;
}


__constant__ char hex2str_map[] = "0123456789ABCDEF";
char* device_msg;
int device_msg_len;
unsigned long long * device_data;
void check(hipError_t ret)
{
    if (ret != hipError_t::hipSuccess)
        printf("cuda failed!\n");
}

__global__ void KGkernel(unsigned long long* data, char* msg_, int len)
{
    int i =blockIdx.x*blockDim.x + threadIdx.x;
    ull pat = data[i];
    ull num = exp_mod(pat, 0x58C5D3F7, 0xF513E783);
    pat = exp_mod(pat, 0xAC3A102B, 0xAE818F1B);
    mul_mod(num, 0x5C4AF104DA37C96D, 0xA70F8F62AA6E97D1);
    mul_mod(pat, 0x4AC49E5DD036CE65, 0xA70F8F62AA6E97D1);
    add_mod(num,pat,0xA70F8F62AA6E97D1);
    uchar msg[64]{ 0 };
    memset(msg, 0, 64);
    *reinterpret_cast<uint*>(&msg[56]) = (len + 16) * 8;
    memcpy(msg, msg_, len);
    msg[len + 16] = 0x80;

    //hex to str start.
    (&msg[len])[0] = hex2str_map[(num >> 15 * 4) & 0xF];
    (&msg[len])[1] = hex2str_map[(num >> 14 * 4) & 0xF];
    (&msg[len])[2] = hex2str_map[(num >> 13 * 4) & 0xF];
    (&msg[len])[3] = hex2str_map[(num >> 12 * 4) & 0xF];
    (&msg[len])[4] = hex2str_map[(num >> 11 * 4) & 0xF];
    (&msg[len])[5] = hex2str_map[(num >> 10 * 4) & 0xF];
    (&msg[len])[6] = hex2str_map[(num >> 9 * 4) & 0xF];
    (&msg[len])[7] = hex2str_map[(num >> 8 * 4) & 0xF];
    (&msg[len])[8] = hex2str_map[(num >> 7 * 4) & 0xF];
    (&msg[len])[9] = hex2str_map[(num >> 6 * 4) & 0xF];
    (&msg[len])[10] = hex2str_map[(num >> 5 * 4) & 0xF];
    (&msg[len])[11] = hex2str_map[(num >> 4 * 4) & 0xF];
    (&msg[len])[12] = hex2str_map[(num >> 3 * 4) & 0xF];
    (&msg[len])[13] = hex2str_map[(num >> 2 * 4) & 0xF];
    (&msg[len])[14] = hex2str_map[(num >> 1 * 4) & 0xF];
    (&msg[len])[15] = hex2str_map[num & 0xF];
    //hex to str end.

    num = md5_512(reinterpret_cast<uint*>(msg));
    data[i] = num;
}

void initKG(char* msg)
{
    device_msg_len = strlen(msg);
    check(hipMalloc(&device_msg, device_msg_len*sizeof(char)));
    check(hipMemcpy(device_msg, msg, device_msg_len*sizeof(char), hipMemcpyHostToDevice));
}

void exitKG()
{
    check(hipFree(device_msg));
}


//the data also hold the calculated data.
//data len is blocks*threads.
void runKGkernel(unsigned long long* data, int blocks, int threads)
{
    check(hipMalloc((void**)&device_data, blocks*threads*sizeof(unsigned long long)));
    check(hipMemcpy(device_data, data, blocks*threads*sizeof(unsigned long long), hipMemcpyHostToDevice));

    KGkernel << <blocks, threads >> >(device_data, device_msg, device_msg_len);
    check(hipMemcpy(data, device_data, blocks*threads*sizeof(unsigned long long), hipMemcpyDeviceToHost));
    check(hipDeviceSynchronize());
    

    check(hipFree(device_data));
}